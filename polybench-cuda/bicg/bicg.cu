/**
 * bicg.c: This file is part of the PolyBench 3.0 test suite.
 *
 *
 * Contact: Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://polybench.sourceforge.net
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

//#define NX 40000
#define NX 40

static unsigned num_blocks(int num, int factor) {
  return (num + factor - 1) / factor;
}



__global__ void kernel_q(int m, int n, double *A, double s[], double q[], double p[], double r[]) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < n) {
    q[i] = 0;
    for (int j = 0; j < m; j++)
      q[i] += A[i * m + j] * p[j];
  }
}


__global__ void kernel_s(int m, int n, double *A, double s[], double q[], double p[], double r[]) {
  int j = blockDim.x * blockIdx.x + threadIdx.x;

  if (j < m) {
    s[j] = 0;
    for (int i = 0; i < n; i++)
      s[j] += r[i] * A[i * m + j];
  }
}


/* Array initialization. */
static
void init_array (int nx, int ny,
		 double *A,
		 double *r,
		 double *p)
{
  int i, j;

  for (i = 0; i < ny; i++)
    p[i] = i * M_PI;
  for (i = 0; i < nx; i++) {
    r[i] = i * M_PI;
    for (j = 0; j < ny; j++)
      A[i*ny+j] = ((double) i*(j+1))/nx;
  }
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int nx, int ny,
		 double *s,
		 double *q)

{
  int i;

  for (i = 0; i < ny; i++) {
    fprintf (stderr, "%0.2lf ", s[i]);
    if (i % 20 == 0) fprintf (stderr, "\n");
  }
  for (i = 0; i < nx; i++) {
    fprintf (stderr, "%0.2lf ", q[i]);
    if (i % 20 == 0) fprintf (stderr, "\n");
  }
  fprintf (stderr, "\n");
}


static void kernel(int m, int n, double *A, double s[], double q[], double p[], double r[]) {
  const unsigned threadsPerBlock = 256;
  kernel_q<<<num_blocks(n, threadsPerBlock), threadsPerBlock>>>(m, n, A, s, q, p, r);
  kernel_s<<<num_blocks(m, threadsPerBlock), threadsPerBlock>>>(m, n, A, s, q, p, r);
}

int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int nx =atoi(argv[2]);
  int ny = atoi(argv[3]);
  int dump_code = atoi(argv[1]);

  /* Variable declaration/allocation. */
  double *A = (double*)malloc(nx*ny*sizeof(double));
  double *s = (double*)malloc(ny*sizeof(double));
  double *q = (double*)malloc(nx*sizeof(double));
  double *p = (double*)malloc(ny*sizeof(double));
  double *r = (double*)malloc(nx*sizeof(double));
  /* Initialize array(s). */
  init_array (nx, ny, A, r, p);

  double *dev_A;
  double *dev_s;
  double *dev_q;
  double *dev_p;
  double *dev_r;
  hipMalloc(&dev_A, nx*ny*sizeof(double));
  hipMalloc(&dev_s, ny*sizeof(double));
  hipMalloc(&dev_q, nx*sizeof(double));
  hipMalloc(&dev_p, ny*sizeof(double));
  hipMalloc(&dev_r, nx*sizeof(double));
  hipMemcpy(dev_A, A, nx*ny*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_s, s, ny*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_q, q, nx*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_p, p, ny*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_r, r, nx*sizeof(double), hipMemcpyHostToDevice);

  kernel(nx, ny, dev_A, dev_s, dev_q, dev_p, dev_r);
  hipMemcpy(s, dev_s, ny*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(q, dev_q, nx*sizeof(double), hipMemcpyDeviceToHost);
  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  if(dump_code == 1) print_array(nx, ny, s, q);

  /* Be clean. */
  free((void*)A);
  free((void*)s);
  free((void*)q);
  free((void*)p);
  free((void*)r);
  hipFree((void*)dev_A);
  hipFree((void*)dev_s);
  hipFree((void*)dev_q);
  hipFree((void*)dev_p);
  hipFree((void*)dev_r);

  return 0;
}
