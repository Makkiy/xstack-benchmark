/**
 * syr2k.c: This file is part of the PolyBench/C 3.2 test suite.
 *
 *
 * Contact: Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://polybench.sourceforge.net
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <math.h>


static unsigned num_blocks(int num, int factor) {
  return (num + factor - 1) / factor;
}


__global__ void kernel_beta(int n, int m,
                            double alpha, double beta,
                            double *C,
                            double *A,
                            double *B) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;


  if (i < n && j <= i)
    C[i * n + j] *= beta;
}

__global__ void kernel_product(int n, int m,
                               double alpha, double beta,
                               double *C,
                               double *A,
                               double *B) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;


  if (i < n && j <= i) {
    for (int k = 0; k < m; k++)
      C[i * n + j] += A[j * m + k] * alpha * B[i * m + k] + B[j * m + k] * alpha * A[i * m + k];
  }
}



static void kernel(int n, int m,
                   double alpha, double beta,
                   double *C,
                   double *A,
                   double *B) {
  const unsigned int threadsPerBlock = 256;

  {
    dim3 block(threadsPerBlock / 32, 32, 1);
    dim3 grid(num_blocks(n, block.x), num_blocks(n, block.y), 1);
    kernel_beta<<<grid, block>>>(n, m, alpha, beta, C, A, B);
  }

  {
    dim3 block(threadsPerBlock / 32, 32, 1);
    dim3 grid(num_blocks(n, block.x), num_blocks(n, block.y), 1);
    kernel_product<<<grid, block>>>(n, m, alpha, beta, C, A, B);
  }
}

  static
void init_array(int ni, int nj,
    double *alpha,
    double *beta,
    double *C,
    double *A,
    double *B)
{
  int i, j;

  *alpha = 32412;
  *beta = 2123;
  for (i = 0; i < ni; i++)
    for (j = 0; j < nj; j++) {
      A[i*nj+j] = ((double) i*j) / ni;
      B[i*nj+j] = ((double) i*j) / ni;
    }
  for (i = 0; i < ni; i++)
    for (j = 0; j < ni; j++)
      C[i*ni+j] = ((double) i*j) / ni;
}




  static
void print_array(int ni,
    double *C)
{
  int i, j;

  for (i = 0; i < ni; i++)
    for (j = 0; j < ni; j++) {
      fprintf (stderr, "%0.2lf ", C[i*ni+j]);
      if ((i * ni + j) % 20 == 0) fprintf (stderr, "\n");
    }
  fprintf (stderr, "\n");
}





int main(int argc, char** argv)
{
  int dump_code = atoi(argv[1]);
  int ni = atoi(argv[2]);
  int nj = atoi(argv[3]);

  double *alpha = (double*)malloc(sizeof(double));
  double *beta = (double*)malloc(sizeof(double));
  double *A = (double*)malloc(ni*nj*sizeof(double));
  double *B = (double*)malloc(ni*nj*sizeof(double));
  double *C = (double*)malloc(ni*ni*sizeof(double));


  init_array (ni, nj, alpha, beta,
      C,
      A,
      B);


  double *dev_A;
  double *dev_B;
  double *dev_C;
  double *dev_alpha;
  double *dev_beta;
  hipMalloc(&dev_A, ni*nj*sizeof(double));
  hipMalloc(&dev_B, ni*nj*sizeof(double));
  hipMalloc(&dev_C, ni*ni*sizeof(double));
  hipMalloc(&dev_alpha, sizeof(double));
  hipMalloc(&dev_beta, sizeof(double));
  hipMemcpy(dev_A, A, ni*nj*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_B, B, ni*nj*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_C, C, ni*ni*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_alpha, alpha, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_beta, beta, sizeof(double), hipMemcpyHostToDevice);


  kernel(ni, nj, *alpha, *beta, dev_C, dev_A, dev_B);
  hipMemcpy(C, dev_C, ni*nj*sizeof(double), hipMemcpyDeviceToHost);




  if (dump_code == 1) print_array(ni, C);


  free((void*)C);;
  free((void*)A);;
  free((void*)B);;
  free((void*)alpha);
  free((void*)beta);

  hipFree((void*)dev_A);
  hipFree((void*)dev_B);
  hipFree((void*)dev_C);
  hipFree((void*)dev_alpha);
  hipFree((void*)dev_beta);

  return 0;
}
